#include "hip/hip_runtime.h"

#include <stdio.h>

void checkDeviceMemory(void) {
	size_t free, total;
	hipMemGetInfo(&free, &total); // build-in function
	printf("Device memory (free/total) = %lld/%lld bytes\n", free, total);
}

int main(void){
	int* dDataPtr; // prefix with d for varirable realted to device (GPU)
	hipError_t errorCode;

	checkDeviceMemory();
	errorCode = hipMalloc(&dDataPtr, sizeof(int) * 1024 * 1024); // allocate memory for 1024 * 1024 int type 
	// errorCode = cudaMalloc(&dDataPtr, sizeof(int) * 1024 * 1024 * 1024 * 8); // error when allcocating memory over free available memory - cudaErrorMemoryAllocation
	printf("hipMalloc - %s\n", hipGetErrorName(errorCode)); // built-in function: cudaGetErrorName()
	checkDeviceMemory();

	errorCode = hipMemset(dDataPtr, 0, sizeof(int) * 1024 * 1024); // initialize garbage value to 0
	printf("hipMemset - %s\n", hipGetErrorName(errorCode));

	errorCode = hipFree(dDataPtr); // free memory
	printf("hipFree - %s\n", hipGetErrorName(errorCode));
	checkDeviceMemory();
}