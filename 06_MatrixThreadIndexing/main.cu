#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define ROW_SIZE 32
#define COL_SIZE 32


// Simple vector sum kernel (Max vector size : 1024)
__global__ void matAdd_2D_index(float* _a, float* _b, float* _c) {
	unsigned int col = threadIdx.x;
	unsigned int row = threadIdx.y;
	unsigned int index = row * blockDim.x + col;

	_c[index] = _a[index] + _b[index];
}

int main(void)
{
	float A[ROW_SIZE][COL_SIZE] = { 0 };
	float B[ROW_SIZE][COL_SIZE] = { 0 };
	float C[ROW_SIZE][COL_SIZE] = { 0 };
	float hC[ROW_SIZE][COL_SIZE] = { 0 };

	for (int iRow = 0; iRow < ROW_SIZE; iRow++) {
		for (int iCol = 0; iCol < COL_SIZE; iCol++) {
			A[iRow][iCol] = rand() % 100;
			B[iRow][iCol] = rand() % 100;
			C[iRow][iCol] = A[iRow][iCol] + B[iRow][iCol];
		}
	}

	int matSize = ROW_SIZE * COL_SIZE;
	float* dA = NULL;
	float* dB = NULL;
	float* dC = NULL;
	hipMalloc(&dA, sizeof(float) * matSize); hipMemset(dA, 0, sizeof(float) * matSize);
	hipMalloc(&dB, sizeof(float) * matSize); hipMemset(dB, 0, sizeof(float) * matSize);
	hipMalloc(&dC, sizeof(float) * matSize); hipMemset(dC, 0, sizeof(float) * matSize);

	hipMemcpy(dA, A, sizeof(float) * matSize, hipMemcpyHostToDevice);
	hipMemcpy(dB, B, sizeof(float) * matSize, hipMemcpyHostToDevice);

	dim3 blockDim(COL_SIZE, ROW_SIZE);
	matAdd_2D_index <<<1, blockDim>>> (dA, dB, dC);

	hipMemcpy(hC, dC, sizeof(float) * matSize, hipMemcpyDeviceToHost);

	// validation
	bool isCorrect = true;
	for (int iRow = 0; iRow < ROW_SIZE; iRow++)
		for (int iCol = 0; iCol < COL_SIZE; iCol++) {
			//printf("%f %f\n", hC[iRow][iCol], C[iRow][iCol]);
			if (hC[iRow][iCol] != C[iRow][iCol]) {
				isCorrect = false;
				break;
			}
		}

	if (isCorrect) printf("GPU works well!\n");
	else printf("GPU fail to make correct result(s)..\n");

	return 0;
}