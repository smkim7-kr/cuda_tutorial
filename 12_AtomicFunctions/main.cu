#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_BLOCK 10240
#define NUM_T_IN_B 512

__global__ void threadCounting_noSync(int* a)
{
    (*a)++;
}

__global__ void threadCounting_atomicGlobal(int* a)
{
    atomicAdd(a, 1);
}

__global__ void threadCounting_atomicShared(int* a)
{
    __shared__ int sa;

    if (threadIdx.x == 0)
        sa = 0;
    __syncthreads();

    atomicAdd(&sa, 1);
    __syncthreads();

    if (threadIdx.x == 0)
        atomicAdd(a, sa);
}

__global__ void threadCounting_warpLvSync(int* a)
{
    __shared__ int wa[NUM_T_IN_B / 32];
    __shared__ int sa;

    int warpID = (int)threadIdx.x / 32;

    if (threadIdx.x % 32 == 0)
        wa[warpID] = 0;
    __syncwarp();

    atomicAdd(&wa[warpID], 1);

    __syncwarp();

    if (threadIdx.x % 32 == 0)
        atomicAdd(&sa, wa[warpID]);

    __syncthreads();

    if (threadIdx.x == 0)
        atomicAdd(a, sa);
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("CUDA Error %s: %s\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

int main(void) {
    int a = 0;
    int* d1, * d2, * d3, * d4;

    hipMalloc((void**)&d1, sizeof(int));
    hipMemset(d1, 0, sizeof(int));

    hipMalloc((void**)&d2, sizeof(int));
    hipMemset(d2, 0, sizeof(int));

    hipMalloc((void**)&d3, sizeof(int));
    hipMemset(d3, 0, sizeof(int));

    hipMalloc((void**)&d4, sizeof(int));
    hipMemset(d4, 0, sizeof(int));

    // CUDA event variables for timing
    hipEvent_t start, stop;
    float elapsedTime;

    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Warm-up kernel
    threadCounting_noSync << <NUM_BLOCK, NUM_T_IN_B >> > (d1);
    hipDeviceSynchronize();

    // ---------------------- No Sync Kernel ----------------------
    hipEventRecord(start, 0);
    threadCounting_noSync << <NUM_BLOCK, NUM_T_IN_B >> > (d1);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(&a, d1, sizeof(int), hipMemcpyDeviceToHost);
    printf("[No Sync.] # of threads = %d\n", a);
    printf("Execution time (No Sync): %f ms\n", elapsedTime);

    // ---------------------- Atomic Global Kernel ----------------------
    hipEventRecord(start, 0);
    threadCounting_atomicGlobal << <NUM_BLOCK, NUM_T_IN_B >> > (d2);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(&a, d2, sizeof(int), hipMemcpyDeviceToHost);
    printf("[AtomicGlobal] # of threads = %d\n", a);
    printf("Execution time (Atomic Global): %f ms\n", elapsedTime);

    // ---------------------- Atomic Shared Kernel ----------------------
    hipEventRecord(start, 0);
    threadCounting_atomicShared << <NUM_BLOCK, NUM_T_IN_B >> > (d3);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(&a, d3, sizeof(int), hipMemcpyDeviceToHost);
    printf("[AtomicShared] # of threads = %d\n", a);
    printf("Execution time (Atomic Shared): %f ms\n", elapsedTime);

    // ---------------------- Warp Level Sync Kernel ----------------------
    hipEventRecord(start, 0);
    threadCounting_warpLvSync << <NUM_BLOCK, NUM_T_IN_B >> > (d4);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    hipMemcpy(&a, d4, sizeof(int), hipMemcpyDeviceToHost);
    printf("[AtomicWarp] # of threads = %d\n", a);
    printf("Execution time (Atomic Warp): %f ms\n", elapsedTime);

    // Free GPU memory
    hipFree(d1);
    hipFree(d2);
    hipFree(d3);
    hipFree(d4);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
