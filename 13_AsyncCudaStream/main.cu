#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_BLOCK (128*1024)
#define NUM_T_IN_B 1024
#define ARRAY_SIZE (NUM_T_IN_B * NUM_BLOCK)
#define NUM_STREAMS 4

__global__ void myKernel(int* _in, int* _out)
{
    int tID = blockDim.x * blockIdx.x + threadIdx.x;

    int temp = 0;
    for (int i = 0; i < 250; i++) {
        temp = (temp + _in[tID] * 5) % 10;
    }
    _out[tID] = temp;
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("CUDA Error %s: %s\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

int main(void)
{
    // Arrays for host and device
    int* in = NULL, * out = NULL, * out2 = NULL;

    // Allocate pinned memory on the host
    checkCudaError(hipHostMalloc(&in, sizeof(int) * ARRAY_SIZE), "hipHostMalloc in");
    checkCudaError(hipHostMalloc(&out, sizeof(int) * ARRAY_SIZE), "hipHostMalloc out");
    checkCudaError(hipHostMalloc(&out2, sizeof(int) * ARRAY_SIZE), "hipHostMalloc out2");

    // Initialize input array with random values
    for (int i = 0; i < ARRAY_SIZE; i++) {
        in[i] = rand() % 10;
    }

    // Allocate memory on the device
    int* dIn, * dOut;
    checkCudaError(hipMalloc(&dIn, sizeof(int) * ARRAY_SIZE), "hipMalloc dIn");
    checkCudaError(hipMalloc(&dOut, sizeof(int) * ARRAY_SIZE), "hipMalloc dOut");

    // CUDA event variables for timing
    hipEvent_t start, stop;
    float elapsedTime;

    // Create CUDA events
    checkCudaError(hipEventCreate(&start), "hipEventCreate start");
    checkCudaError(hipEventCreate(&stop), "hipEventCreate stop");

    // ------------------- Single Stream Version -------------------
    printf("Starting Single Stream Version...\n");

    // Start timing
    hipEventRecord(start, 0);

    // Transfer data from host to device
    checkCudaError(hipMemcpy(dIn, in, sizeof(int) * ARRAY_SIZE, hipMemcpyHostToDevice), "hipMemcpy HtoD");

    // Launch the kernel
    myKernel << <NUM_BLOCK, NUM_T_IN_B >> > (dIn, dOut);
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");

    // Transfer data from device to host
    checkCudaError(hipMemcpy(out, dOut, sizeof(int) * ARRAY_SIZE, hipMemcpyDeviceToHost), "hipMemcpy DtoH");

    // Stop timing
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Single stream execution time: %f ms\n", elapsedTime);

    // ------------------- Multiple Streams Version -------------------
    printf("Starting Multiple Streams Version...\n");

    // Create streams
    hipStream_t stream[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaError(hipStreamCreate(&stream[i]), "hipStreamCreate");
    }

    // Start timing
    hipEventRecord(start, 0);

    int chunkSize = ARRAY_SIZE / NUM_STREAMS;

    // Launch async operations with multiple streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = chunkSize * i;
        checkCudaError(hipMemcpyAsync(dIn + offset, in + offset, sizeof(int) * chunkSize, hipMemcpyHostToDevice, stream[i]), "hipMemcpyAsync HtoD");
        myKernel << <NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i] >> > (dIn + offset, dOut + offset);
        checkCudaError(hipMemcpyAsync(out2 + offset, dOut + offset, sizeof(int) * chunkSize, hipMemcpyDeviceToHost, stream[i]), "hipMemcpyAsync DtoH");
    }

    // Synchronize all streams
    checkCudaError(hipDeviceSynchronize(), "hipDeviceSynchronize");

    // Stop timing
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    printf("Multiple streams execution time: %f ms\n", elapsedTime);

    // ------------------- Compare Results -------------------
    for (int i = 0; i < ARRAY_SIZE; i++) {
        if (out[i] != out2[i]) {
            printf("Mismatch at index %d: Single Stream = %d, Multiple Streams = %d\n", i, out[i], out2[i]);
            break;
        }
    }

    // Clean up
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaError(hipStreamDestroy(stream[i]), "hipStreamDestroy");
    }

    hipFree(dIn);
    hipFree(dOut);
    hipHostFree(in);
    hipHostFree(out);
    hipHostFree(out2);

    // Destroy CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
