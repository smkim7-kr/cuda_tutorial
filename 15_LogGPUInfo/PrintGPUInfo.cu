#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime_api.h"

#include <stdio.h>

#define _1MB (1024*1024)

void main(void) {
	int ngpus;
	hipGetDeviceCount(&ngpus); // save number of gpus into variable ngpus

	for (int i = 0; i < ngpus; i++) {
		hipDeviceProp_t devProp; // cudaDevieProp struct

		hipGetDeviceProperties(&devProp, i);

		printf("Device %d: %s\n", i, devProp.name);
		printf("\tCompute capability: %d.%d\n", devProp.major, devProp.minor);
		printf("\tThe number of streaming multiprocessors: %d\n", devProp.multiProcessorCount);
		printf("\tThe number of CUDA cores: %d\n", _ConvertSMVer2Cores(devProp.major, devProp.minor) * devProp.multiProcessorCount);
		printf("\tGlobal memory size: %.2f MB\n", (float)devProp.totalGlobalMem / _1MB);
	}

	hipSetDevice(0);
	int gpuid = 10;
	hipGetDevice(&gpuid);
	printf("Current running gpu is gpu(%d)\n", gpuid);
	
}