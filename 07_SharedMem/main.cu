#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#define ROW_SIZE (32)
#define K_SIZE   (128)
#define COL_SIZE (32)
#define WORK_LOAD (4096) // access same matrix multiplication multiple times - to emphsize the effect of shared memory
#define MAT_SIZE_A (ROW_SIZE*K_SIZE)
#define MAT_SIZE_B (K_SIZE*COL_SIZE)
#define MAT_SIZE_C (ROW_SIZE*COL_SIZE)

// Input matrices
float A[ROW_SIZE][K_SIZE];  // m * k
float B[K_SIZE][COL_SIZE];  // k * n

// Kernel for matrix multiplication (standard)
__global__ void matMul_kernel(float* _A, float* _B, float* _C) {
    int row = threadIdx.x;
    int col = threadIdx.y;
    int index = row * blockDim.y + col;

    float result = 0;
    for (int k = 0; k < K_SIZE; k++)
        for (int i = 0; i < WORK_LOAD; i++)
            result += _A[row * K_SIZE + k] * _B[col + k * COL_SIZE];
    _C[index] = result;
}

// Kernel for matrix multiplication with shared memory
__global__ void matMul_kernel_shared_master_init(float* _A, float* _B, float* _C) {
    int row = threadIdx.x;
    int col = threadIdx.y;
    int index = row * blockDim.y + col;

    __shared__ float sA[ROW_SIZE][K_SIZE];  // Shared memory for A
    __shared__ float sB[K_SIZE][COL_SIZE];  // Shared memory for B

    if (threadIdx.x == 0 && threadIdx.y == 0) {
        for (int r = 0; r < ROW_SIZE; r++) {
            for (int k = 0; k < K_SIZE; k++) {
                sA[r][k] = _A[r * K_SIZE + k];
            }
        }
        for (int c = 0; c < COL_SIZE; c++) {
            for (int k = 0; k < K_SIZE; k++) {
                sB[k][c] = _B[c + k * COL_SIZE];
            }
        }
    }

    __syncthreads();  // Ensure all threads load the data

    float result = 0;
    for (int k = 0; k < K_SIZE; k++)
        for (int i = 0; i < WORK_LOAD; i++)
            result += sA[row][k] * sB[k][col];
    _C[index] = result;
}

int main(void) {
    float* dA, * dB, * dC;
    hipMalloc(&dA, sizeof(float) * MAT_SIZE_A);
    hipMalloc(&dB, sizeof(float) * MAT_SIZE_B);
    hipMalloc(&dC, sizeof(float) * MAT_SIZE_C);

    // Initialize input matrices (A and B) with random values
    for (int r = 0; r < ROW_SIZE; r++)
        for (int k = 0; k < K_SIZE; k++)
            A[r][k] = rand() % 100;

    for (int k = 0; k < K_SIZE; k++)
        for (int c = 0; c < COL_SIZE; c++)
            B[k][c] = rand() % 100;

    // Copy input matrices to device
    hipMemcpy(dA, A, sizeof(float) * MAT_SIZE_A, hipMemcpyHostToDevice);
    hipMemcpy(dB, B, sizeof(float) * MAT_SIZE_B, hipMemcpyHostToDevice);

    // Timing variables for CUDA events
    hipEvent_t start, stop;
    float kernel_time = 0.0f, kernel_shared_time = 0.0f;

    // Create CUDA events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    dim3 blockDim(ROW_SIZE, COL_SIZE);

    // Measure time for standard kernel
    hipEventRecord(start, 0);
    matMul_kernel << <1, blockDim >> > (dA, dB, dC);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernel_time, start, stop);

    // Measure time for shared memory kernel
    hipEventRecord(start, 0);
    matMul_kernel_shared_master_init << <1, blockDim >> > (dA, dB, dC);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&kernel_shared_time, start, stop);

    // Release CUDA events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // Free device memory
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // Print timing results
    printf("Kernel execution time (standard): %f ms\n", kernel_time);
    printf("Kernel execution time (shared memory): %f ms\n", kernel_shared_time);

    return 0;
}
