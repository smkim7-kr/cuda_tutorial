#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define NUM_BLOCK (128*1024)
#define NUM_T_IN_B 1024
#define ARRAY_SIZE (NUM_T_IN_B * NUM_BLOCK)
#define NUM_STREAMS 4

__global__ void myKernel(int* _in, int* _out)
{
    int tID = blockDim.x * blockIdx.x + threadIdx.x;

    int temp = 0;
    for (int i = 0; i < 250; i++) {
        temp = (temp + _in[tID] * 5) % 10;
    }
    _out[tID] = temp;
}

void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        printf("CUDA Error %s: %s\n", msg, hipGetErrorString(err));
        exit(-1);
    }
}

int main(void)
{
    // Arrays for host and device
    int* in = NULL, * out = NULL;

    // Allocate pinned memory on the host
    checkCudaError(hipHostMalloc(&in, sizeof(int) * ARRAY_SIZE), "hipHostMalloc in");
    checkCudaError(hipHostMalloc(&out, sizeof(int) * ARRAY_SIZE), "hipHostMalloc out");

    // Initialize input array with random values
    for (int i = 0; i < ARRAY_SIZE; i++) {
        in[i] = rand() % 10;
    }

    // Allocate memory on the device
    int* dIn, * dOut;
    checkCudaError(hipMalloc(&dIn, sizeof(int) * ARRAY_SIZE), "hipMalloc dIn");
    checkCudaError(hipMalloc(&dOut, sizeof(int) * ARRAY_SIZE), "hipMalloc dOut");

    // Create streams
    hipStream_t stream[NUM_STREAMS];
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaError(hipStreamCreate(&stream[i]), "hipStreamCreate");
    }

    // CUDA event variables for timing for each stream
    hipEvent_t start[NUM_STREAMS], stop[NUM_STREAMS];
    float elapsedTime[NUM_STREAMS];

    // Create CUDA events for each stream
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaError(hipEventCreate(&start[i]), "hipEventCreate start");
        checkCudaError(hipEventCreate(&stop[i]), "hipEventCreate stop");
    }

    int chunkSize = ARRAY_SIZE / NUM_STREAMS;

    // Launch async operations with multiple streams and measure time
    for (int i = 0; i < NUM_STREAMS; i++) {
        int offset = chunkSize * i;

        // Record the start event for this stream
        checkCudaError(hipEventRecord(start[i], stream[i]), "hipEventRecord start");

        // Async memory copy and kernel execution
        checkCudaError(hipMemcpyAsync(dIn + offset, in + offset, sizeof(int) * chunkSize, hipMemcpyHostToDevice, stream[i]), "hipMemcpyAsync HtoD");
        myKernel << <NUM_BLOCK / NUM_STREAMS, NUM_T_IN_B, 0, stream[i] >> > (dIn + offset, dOut + offset);
        checkCudaError(hipMemcpyAsync(out + offset, dOut + offset, sizeof(int) * chunkSize, hipMemcpyDeviceToHost, stream[i]), "hipMemcpyAsync DtoH");

        // Record the stop event for this stream
        checkCudaError(hipEventRecord(stop[i], stream[i]), "hipEventRecord stop");
    }

    // Synchronize streams and measure elapsed time
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaError(hipStreamSynchronize(stream[i]), "hipStreamSynchronize");
        checkCudaError(hipEventElapsedTime(&elapsedTime[i], start[i], stop[i]), "hipEventElapsedTime");
    }

    // Print execution times for each stream
    for (int i = 0; i < NUM_STREAMS; i++) {
        printf("Stream %d execution time: %f ms\n", i, elapsedTime[i]);
    }

    // Clean up
    for (int i = 0; i < NUM_STREAMS; i++) {
        checkCudaError(hipStreamDestroy(stream[i]), "hipStreamDestroy");
        checkCudaError(hipEventDestroy(start[i]), "hipEventDestroy start");
        checkCudaError(hipEventDestroy(stop[i]), "hipEventDestroy stop");
    }

    hipFree(dIn);
    hipFree(dOut);
    hipHostFree(in);
    hipHostFree(out);

    return 0;
}
